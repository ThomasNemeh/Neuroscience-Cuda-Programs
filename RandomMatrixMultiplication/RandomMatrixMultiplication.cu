//Program to multiply square matrix with activation vector, both filled with random numbers, and then to multiply the resulting vector by the matrix again- repeat for the 
//specified number of iterations.

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cstdlib>
#include <iostream>
using std::cout;
using std::endl;
using std::copy;

float *h_A = nullptr;
float *h_B = nullptr;

// fills matrix with random float 
// Param: pointer to matrix, number of rows, number of columns
void GPU_fill_rand(float *matrix, int rows, int cols) {
     // Create a pseudo-random number generator
     hiprandGenerator_t prng;
     hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
 
     // Set the seed for the random number generator using the system clock
     hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
 
     // Fill the array with uniformly distributed random numbers on the device between 0 and 1, where 0 is included and 1 is excluded
      hiprandGenerateUniform(prng, matrix, rows * cols);
}

// converts float values to integer between 0 and 10, where 0 is included and 10 is excluded
// Param: pointer to matrix, number of elements in matrix
__global__ void changeValues(float *matrix, int size) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < size) {
		float a = matrix[index] * 10;
		int b = (int) a;
		matrix[index] = (float) b;
		
	}
}

//Print matrix storage in column-major format
//Param: pointer to matrix, number of rows, number of columns
void print_matrix(const float *matrix, int rows, int cols) {
 
    for(int i = 0; i < rows; ++i){
        for(int j = 0; j < cols; ++j){
            std::cout << matrix[j * rows + i] << " ";
        }
        std::cout << std::endl;
	}
    std::cout << std::endl;
}

// perform the sqeeze function on each element of the vector resulting from the later iteration of matrix multiplication
// Param: B = pointer to activation vector, dim = starting point of the vector results of the last iteration of matrix multiplication, 
// L and M are parameters of the squeeze function
__global__ void squeeze(float *B, int dim, int length, float L, float M) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
	if (index < length + dim) {
		B[index] = 1 / (1 + expf(-1 * L * (B[index] - M)));
	}
}

// perform the matrix multiplication operation
// Param: handle = handle to the cuBLAS library context. iterations = number of times we multiply activation vector by matrix
//        A = matrix. B = array of activation vectors calculated so far. dim = length & width of square matrix. L, M = parameter for squeeze function
void gpu_blas_mmul(hipblasHandle_t &handle, int iterations, const float *A, float *B, const int dim, float L, float M) {
    const float alf = 1; // scalar used for multiplication
    const float bet = 0; // scalar used for multiplication
    const float *alpha = &alf;
    const float *beta = &bet;
	int length = 0;
	
	for (int i = 0; i < iterations; i++) {
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, 1, dim, alpha, A, dim, B, dim, beta, (B + length), dim);
		squeeze<<<(31 + dim) / 32, 32>>>(B, dim, length, L, M);
		length += dim;
	} 
}

// external function defined in RandomMatrixMultiplication.h
extern "C++" float *matrixMultiplication(int dim, int iterations, float L, float M) {
	int size_A = dim * dim;
	
	// allocate square matrix on host
	h_A = new float[size_A];
	//allocate array to hold activation vectors on host
    h_B = new float[dim * iterations];
	
	// allocate arrays on device
	float *dev_A, *dev_B;
	hipMalloc(&dev_A, size_A * sizeof(float));
	hipMalloc(&dev_B, dim * iterations * sizeof(float));
	
	// fill matrix and first activation vector with random values
	GPU_fill_rand(dev_A, dim, dim);
    GPU_fill_rand(dev_B, dim, 1);
	
	// change decimal values in matrix to integers between 0 and 10
	changeValues<<<(31 + size_A) / 32, 32>>>(dev_A, size_A);
	
	// create handle to the cuBLAS library context
	hipblasHandle_t handle;
    hipblasCreate(&handle);
	
	gpu_blas_mmul(handle, iterations, dev_A, dev_B, dim, L, M);
	
	// destroy handle
	hipblasDestroy(handle);
	
	// copy results to host
	hipMemcpy(h_A, dev_A, size_A * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_B, dev_B, dim * iterations * sizeof(float),hipMemcpyDeviceToHost);

    //Free GPU memory
    hipFree(dev_A);
    hipFree(dev_B);
	
	return h_B;
    
}

// return matrix
extern "C++" float *getMatrix() {
	return h_A;
}