//Program to multiply square matrix with activation vector, both filled with random numbers, and then to multiply the resulting vector by the matrix again- repeat for the 
//specified number of iterations.

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cstdlib>
#include <iostream>
using std::cout;
using std::endl;
using std::copy;

// basic file operations
#include <iostream>
#include <fstream>
using namespace std;

float *h_A = nullptr;
float *h_B = nullptr;

// fills matrix with random float 
// Param: pointer to matrix, number of rows, number of columns
void GPU_fill_rand(float *matrix, int rows, int cols) {
     // Create a pseudo-random number generator
     hiprandGenerator_t prng;
     hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
 
     // Set the seed for the random number generator using the system clock
     hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
 
     // Fill the array with uniformly distributed random numbers on the device between 0 and 1, where 0 is included and 1 is excluded
      hiprandGenerateUniform(prng, matrix, rows * cols);
}

// converts float values to integer between 0 and 10, where 0 is included and 10 is excluded
// Param: pointer to matrix, number of elements in matrix
__global__ void changeValues(float *matrix, int size) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < size) {
		float a = matrix[index] * 10;
		int b = (int) a;
		matrix[index] = (float) b;
		
	}
}

//Print matrix storage in column-major format
//Param: pointer to matrix, number of rows, number of columns
void print_matrix(float *matrix, int rows, int cols) {
 
    for(int i = 0; i < rows; ++i){
        for(int j = 0; j < cols; ++j){
            //std::cout << matrix[j * rows + i] << " ";
			matrix[j * rows + i] = 0;
        }
        //std::cout << std::endl;
	}
    //std::cout << std::endl;
}

// perform the sqeeze function on each element of the vector resulting from the later iteration of matrix multiplication
// Param: B = pointer to activation vector, dim = starting point of the vector results of the last iteration of matrix multiplication, 
// L and M are parameters of the squeeze function
__global__ void updateState(float *B, float *external, int dim, float timestep, int length, float L, float M) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
	if (index < length + dim) {
		float input = B[index] + external[index-dim];
		float old_output = B[index - dim];
		float d_layers = (-1 * old_output) + 1 / (1 + expf(-1 * L * (input - M)));
		B[index] = old_output + d_layers * timestep;
	}
}

// perform the matrix multiplication operation
// Param: handle = handle to the cuBLAS library context. iterations = number of times we multiply activation vector by matrix
//        A = matrix. B = array of activation vectors calculated so far. dim = length & width of square matrix. L, M = parameter for squeeze function
void gpu_blas_mmul(hipblasHandle_t &handle, int iterations, float timestep, const float *A, float *B, float *external, const int dim, float L, float M) {
    const float alf = 1; // scalar used for multiplication
    const float bet = 0; // scalar used for multiplication
    const float *alpha = &alf;
    const float *beta = &bet;
	int length = dim;
	
	for (int i = 0; i < iterations; i++) {
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dim, 1, dim, alpha, A, dim, (B + length - dim), dim, beta, (B + length), dim);
		updateState<<<(31 + dim) / 32, 32>>>(B, external, dim, timestep, length, L, M);
		length += dim;
	} 
}

extern "C++" void fillWeights(float *weights, int dim) {
	// allocate weight connections matrix on the GPU
	float *dev_weights;
	int size_weights = dim * dim;
	hipMalloc(&dev_weights, size_weights * sizeof(float));
	
	// fill matrix and first activation vector with random values
	GPU_fill_rand(dev_weights, dim, dim);
	
	// change decimal values in matrix to integers between 0 and 10
	changeValues<<<(31 + size_weights) / 32, 32>>>(dev_weights, size_weights);
	
	// copy results to host
	hipMemcpy(weights, dev_weights, size_weights * sizeof(float), hipMemcpyDeviceToHost);
	
	//Free GPU memory
    hipFree(dev_weights);
}

extern "C++" void fillLayers(float *layers, int dim) {
	// allocate neurons vector on the GPU
	float *dev_layers;
	int size_layers = dim;
	hipMalloc(&dev_layers, size_layers * sizeof(float));
	
	// fill vector vector with random values
	GPU_fill_rand(dev_layers, dim, 1);
	
	// copy results to host
	hipMemcpy(layers, dev_layers, dim * sizeof(float), hipMemcpyDeviceToHost);
	
	//Free GPU memory
    hipFree(dev_layers);
}

// external function defined in MatrixMultiplication.h
extern "C++" void matrixMultiplication(float *layers, float *weights, float *external, int dim, int iterations, float timestep, float L, float M) {
	int size_weights = dim * dim;
	int size_layers = dim * iterations + dim;

	// allocate arrays on device
	float *dev_layers, *dev_weights, *dev_external;
	hipMalloc(&dev_layers, size_layers * sizeof(float));
	hipMalloc(&dev_external, size_layers * sizeof(float));
	hipMalloc(&dev_weights, size_weights * sizeof(float));
	
	// copy arrays to GPU
	hipMemcpy(dev_layers, layers, size_layers * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_external, external, size_layers * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_weights, weights, size_weights * sizeof(float),hipMemcpyHostToDevice);
	
	// create handle to the cuBLAS library context
	hipblasHandle_t handle;
    hipblasCreate(&handle);
	
	gpu_blas_mmul(handle, iterations, timestep, dev_weights, dev_layers, dev_external, dim, L, M);
	
	// destroy handle
	hipblasDestroy(handle);
	
	// copy results to host
	hipMemcpy(layers, dev_layers, size_layers * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(external, dev_external, size_layers * sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(weights, dev_weights, size_weights * sizeof(float),hipMemcpyDeviceToHost);

    //Free GPU memory
    hipFree(dev_layers);
    hipFree(dev_weights);
	hipFree(dev_external);
    
}