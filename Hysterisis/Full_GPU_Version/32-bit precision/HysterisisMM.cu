//Program to multiply square matrix with activation vector, both filled with random numbers, and then to multiply the resulting vector by the matrix again- repeat for the
//specified number of iterations.

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdlib>
#include <iostream>
using std::cout;
using std::endl;
using std::copy;

using namespace std;

// fills matrix with random float
// Param: pointer to matrix, number of rows, number of columns
void GPU_fill_rand(float *matrix, int rows, int cols) {
     // Create a pseudo-random number generator
     hiprandGenerator_t prng;
     hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

     // Set the seed for the random number generator using the system clock
     hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

     // Fill the array with uniformly distributed random numbers on the device between 0 and 1, where 0 is included and 1 is excluded
      hiprandGenerateUniform(prng, matrix, rows * cols);
}

// converts float values to integer between 0 and 10, where 0 is included and 10 is excluded
// Param: pointer to matrix, number of elements in matrix
__global__ void changeValues(float *matrix, int size) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < size) {
		float a = matrix[index] * 10;
		int b = (int) a;
		matrix[index] = (float) b;

	}
}

//Print matrix storage in column-major format
//Param: pointer to matrix, number of rows, number of columns
void print_matrix(float *matrix, int rows, int cols) {

    for(int i = 0; i < rows; ++i){
        for(int j = 0; j < cols; ++j){
            //std::cout << matrix[j * rows + i] << " ";
			matrix[j * rows + i] = 0;
        }
        //std::cout << std::endl;
	}
    //std::cout << std::endl;
}

// perform the sqeeze function on each element of the vector resulting from the later iteration of matrix multiplication
// Param: B = pointer to activation vector, dim = starting point of the vector results of the last iteration of matrix multiplication,
// L and M are parameters of the squeeze function
__global__ void updateState(float *B, float *external, int dim, float timestep, float noise, int length, int totalIterations, int iterationNum, float L, float M) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
	if (index >= length && index < length + dim) {
		int neuronNum = index % dim;
		float input = B[index] + external[neuronNum * (totalIterations) + iterationNum];
		float old_output = B[index - dim];
		float d_layers = (-1 * old_output) + 1 / (1 + expf(-1 * L * (input - M)));

		// create random number generator
		hiprandState_t state;
		hiprand_init (blockIdx.x * 1000 + threadIdx.x + clock64(), 0, 0, &state);
		float random = hiprand_normal(&state);
		float guassian_noise = noise * random * sqrt(timestep);
		B[index] = old_output + d_layers * timestep + guassian_noise;
	}
}

// perform the matrix multiplication operation
// Param: handle = handle to the cuBLAS library context. iterations = number of times we multiply activation vector by matrix
//        A = matrix. B = array of activation vectors calculated so far. dim = length & width of square matrix. L, M = parameter for squeeze function
void gpu_blas_mmul(hipblasHandle_t &handle, int iterations, float timestep, const float noise, const float *A, float *B, float *external, const int dim, const int size_layers, float L, float M) {
    const float alf = 1; // scalar used for multiplication
    const float bet = 0; // scalar used for multiplication
    const float *alpha = &alf;
    const float *beta = &bet;
	int length = dim;

	for (int i = 0; i < iterations; i++) {
		hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, dim, 1, dim, alpha, A, dim, (B + length - dim), dim, beta, (B + length), dim);
		updateState<<<(31 + dim) / 32, 32>>>(B, external, dim, timestep, noise, length, iterations, i, L, M);
		length += dim;
	}
}

extern "C++" void fillWeights(float *weights, int dim) {
	// allocate weight connections matrix on the GPU
	float *dev_weights;
	int size_weights = dim * dim;
	hipMalloc(&dev_weights, size_weights * sizeof(float));

	// fill matrix and first activation vector with random values
	GPU_fill_rand(dev_weights, dim, dim);

	// change decimal values in matrix to integers between 0 and 10
	changeValues<<<(31 + size_weights) / 32, 32>>>(dev_weights, size_weights);

	// copy results to host
	hipMemcpy(weights, dev_weights, size_weights * sizeof(float), hipMemcpyDeviceToHost);

	//Free GPU memory
    hipFree(dev_weights);
}

extern "C++" void fillLayers(float *layers, int dim) {
	// allocate neurons vector on the GPU
	float *dev_layers;
	int size_layers = dim;
	hipMalloc(&dev_layers, size_layers * sizeof(float));

	// fill vector vector with random values
	GPU_fill_rand(dev_layers, dim, 1);

	// copy results to host
	hipMemcpy(layers, dev_layers, dim * sizeof(float), hipMemcpyDeviceToHost);

	//Free GPU memory
    hipFree(dev_layers);
}

// external function defined in MatrixMultiplication.h
extern "C++" void matrixMultiplication(float *layers, float *weights, float *external, int dim, int iterations, float timestep, float noise, float L, float M) {
	int size_weights = dim * dim;
	int size_layers = dim * iterations + dim;
	int size_external = size_layers * dim - dim;

	// allocate arrays on device
	float *dev_layers, *dev_weights, *dev_external;
	hipMalloc(&dev_layers, size_layers * sizeof(float));
	hipMalloc(&dev_external, size_external * sizeof(float));
	hipMalloc(&dev_weights, size_weights * sizeof(float));

	// copy arrays to GPU
	hipMemcpy(dev_layers, layers, size_layers * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_external, external, size_external * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_weights, weights, size_weights * sizeof(float),hipMemcpyHostToDevice);

	// create handle to the cuBLAS library context
	hipblasHandle_t handle;
    hipblasCreate(&handle);

	gpu_blas_mmul(handle, iterations, timestep, noise, dev_weights, dev_layers, dev_external, dim, size_layers, L, M);

	// destroy handle
	hipblasDestroy(handle);

	// copy results to host
	hipMemcpy(layers, dev_layers, size_layers * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(external, dev_external, size_layers * sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(weights, dev_weights, size_weights * sizeof(float),hipMemcpyDeviceToHost);

    //Free GPU memory
    hipFree(dev_layers);
    hipFree(dev_weights);
	hipFree(dev_external);

}
