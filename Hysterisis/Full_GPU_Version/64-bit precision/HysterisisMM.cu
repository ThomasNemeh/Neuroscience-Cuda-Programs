//Program to multiply square matrix with activation vector, both filled with random numbers, and then to multiply the resulting vector by the matrix again- repeat for the
//specified number of iterations.

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdlib>
#include <iostream>
using std::cout;
using std::endl;
using std::copy;

using namespace std;


// fills matrix with random float
// Param: pointer to matrix, number of rows, number of columns
void GPU_fill_rand(double *matrix, int rows, int cols) {
     // Create a pseudo-random number generator
     hiprandGenerator_t prng;
     hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

     // Set the seed for the random number generator using the system clock
     hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

     // Fill the array with uniformly distributed random numbers on the device between 0 and 1, where 0 is included and 1 is excluded
      hiprandGenerateUniformDouble(prng, matrix, rows * cols);
}

// converts float values to integer between 0 and 10, where 0 is included and 10 is excluded
// Param: pointer to matrix, number of elements in matrix
__global__ void changeValues(double *matrix, int size) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < size) {
		double a = matrix[index] * 10;
		int b = (int) a;
		matrix[index] = (double) b;

	}
}

//Print matrix storage in column-major format
//Param: pointer to matrix, number of rows, number of columns
void print_matrix(double *matrix, int rows, int cols) {

    for(int i = 0; i < rows; ++i){
        for(int j = 0; j < cols; ++j){
            //std::cout << matrix[j * rows + i] << " ";
			matrix[j * rows + i] = 0;
        }
        //std::cout << std::endl;
	}
    //std::cout << std::endl;
}

// perform the sqeeze function on each element of the vector resulting from the later iteration of matrix multiplication
// Param: B = pointer to activation vector, dim = starting point of the vector results of the last iteration of matrix multiplication,
// L and M are parameters of the squeeze function
__global__ void updateState(double *B, double *external, double *lamBeta, int dim, float timestep, double noise, int length, int totalIterations, int iterationNum) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
	if (index >= length && index < length + dim) {
    int neuronNum = index % dim;
    double lam = lamBeta[neuronNum * 2];
    double beta = lamBeta[neuronNum * 2 + 1];

		double input = B[index] + external[neuronNum * (totalIterations) + iterationNum];
		double old_output = B[index - dim];
		double d_layers = (-1 * old_output) + 1 / (1 + expf(-1 * lam * (input - beta)));

		// create random number generator
		hiprandState_t state;
		hiprand_init (blockIdx.x * 1000 + threadIdx.x + clock64(), 0, 0, &state);
		float random = hiprand_normal(&state);
		double guassian_noise = noise * random * sqrt(timestep);
		B[index] = old_output + d_layers * timestep + guassian_noise;
	}
}

// perform the matrix multiplication operation
// Param: handle = handle to the cuBLAS library context. iterations = number of times we multiply activation vector by matrix
// A = matrix. B = array of activation vectors calculated so far. dim = length & width of square matrix. L, M = parameter for squeeze function
void gpu_blas_mmul(hipblasHandle_t &handle, int iterations, float timestep, const double noise, const double *A, double *B, double *external, double *lamBeta, const int dim, const int size_layers) {
    const double alf = 1; // scalar used for multiplication
    const double bet = 0; // scalar used for multiplication
    const double *alpha = &alf;
    const double *beta = &bet;
	  int length = dim;

	for (int i = 0; i < iterations; i++) {
		hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, dim, 1, dim, alpha, A, dim, (B + length - dim), dim, beta, (B + length), dim);
		updateState<<<(31 + dim) / 32, 32>>>(B, external, lamBeta, dim, timestep, noise, length, iterations, i);
		length += dim;
	}
}


extern "C++" void fillWeights(double *weights, int dim) {
	// allocate weight connections matrix on the GPU
	double *dev_weights;
	int size_weights = dim * dim;
	hipMalloc(&dev_weights, size_weights * sizeof(double));

	// fill matrix and first activation vector with random values
	GPU_fill_rand(dev_weights, dim, dim);

	// change decimal values in matrix to integers between 0 and 10
	changeValues<<<(31 + size_weights) / 32, 32>>>(dev_weights, size_weights);

	// copy results to host
	hipMemcpy(weights, dev_weights, size_weights * sizeof(double), hipMemcpyDeviceToHost);

	//Free GPU memory
  hipFree(dev_weights);
}

extern "C++" void fillLayers(double *layers, int dim) {
	// allocate neurons vector on the GPU
	double *dev_layers;
	int size_layers = dim;
	hipMalloc(&dev_layers, size_layers * sizeof(double));

	// fill vector vector with random values
	GPU_fill_rand(dev_layers, dim, 1);

	// copy results to host
	hipMemcpy(layers, dev_layers, dim * sizeof(double), hipMemcpyDeviceToHost);

	//Free GPU memory
  hipFree(dev_layers);
}


// external function defined in MatrixMultiplication.h
extern "C++" void matrixMultiplication(double *layers, double *weights, double *external, double *lamBeta, int dim, int iterations, float timestep, double noise) {
	int size_weights = dim * dim;
	int size_layers = dim * iterations + dim;
	int size_external = size_layers * dim - dim;
  int size_lamBeta = dim * 2;

	// allocate arrays on device
	double *dev_layers, *dev_weights, *dev_external, *dev_lamBeta;
	hipMalloc(&dev_layers, size_layers * sizeof(double));
	hipMalloc(&dev_external, size_external * sizeof(double));
	hipMalloc(&dev_weights, size_weights * sizeof(double));
  hipMalloc(&dev_lamBeta, dim * 2 * sizeof(double));

	// copy arrays to GPU
	hipMemcpy(dev_layers, layers, size_layers * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_external, external, size_external * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_weights, weights, size_weights * sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(dev_lamBeta, lamBeta, size_lamBeta * sizeof(double),hipMemcpyHostToDevice);

	// create handle to the cuBLAS library context
	hipblasHandle_t handle;
  hipblasCreate(&handle);

	gpu_blas_mmul(handle, iterations, timestep, noise, dev_weights, dev_layers, dev_external, dev_lamBeta, dim, size_layers);

	// destroy handle
	hipblasDestroy(handle);

	// copy results to host
	hipMemcpy(layers, dev_layers, size_layers * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(external, dev_external, size_layers * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(weights, dev_weights, size_weights * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(lamBeta, dev_lamBeta, size_lamBeta * sizeof(double), hipMemcpyDeviceToHost);

  //Free GPU memory
  hipFree(dev_layers);
  hipFree(dev_weights);
	hipFree(dev_external);
  hipFree(dev_lamBeta);
}
