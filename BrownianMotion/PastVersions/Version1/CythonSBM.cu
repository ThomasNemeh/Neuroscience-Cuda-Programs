#include "hip/hip_runtime.h"
#include <Python.h>
#include <numpy/arrayobject.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

// Error handling code used in Nvidia example found here: https://docs.nvidia.com/cuda/hiprand/host-api-overview.html#generator-options
#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

//Function to generate brownian path, which is stored in results. Executes on the GPU, hence the __global__ identifier
__global__ void randomWalk(double *results, int T, int N) {
	hiprandState_t state;
	hiprand_init (1234, 0, 0, &state);
	double random;
	
	results[0] = 0.0;
	
	for (int j = 1; j < N; j++) {
		random = hiprand_normal_double(&state);
		results[j] = results[j-1] + random * sqrt((double) T / N);
	}
	
	/*
	Generate 2 doubles at once. Test later to see if this is more efficient:
	double hiprand_normal2_double (state);
	*/
	
}

extern "C++" double *makePath(int T, int N) {
	//Arrays to store the brownian path, one for the host and one for the device
	double *results = new double[N];
	double *dev_results;
	
	// Allocate space for results array on device
	hipMalloc(&dev_results, N * sizeof(double));
	
	//Call GPU function, with ony one block and one thread
	randomWalk<<<1, 1>>>(dev_results, T, N);
	
	//copy results array from device to host
	hipMemcpy(results, dev_results , N * sizeof(double), hipMemcpyDeviceToHost);
	
	//clean up
	hipFree(dev_results);
	
	return results;

}


